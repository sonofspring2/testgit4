#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(float *A, float *B, float *C, int numElements)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}



int main()
{
	int numElements = 1000;
	size_t size = numElements * sizeof(float);

	float *h_A, *h_B, *h_C;

	h_A = (float *) malloc(size);
	h_B = (float *) malloc(size);


	for(int i=0; i < numElements; i++)
	{
		h_A[i] = 1;
		h_B[i] = 2;
	}




	float * d_A = NULL;
	float * d_B = NULL;

	hipMalloc( (void **)&d_A, size );
	hipMalloc( (void **)&d_B, size );

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


	float * d_C = NULL;

	hipMalloc( (void **)&d_C, size  );



}